#include "hip/hip_runtime.h"
/*
 * caesar.cu
 *
 * You can implement your CUDA-accelerated encryption and decryption algorithms
 * in this file.
 *
 */

#include "file.hh"
#include "timer.hh"
#include <cctype>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

using namespace std;

#define MAX_KEY_LENGTH 256
__constant__ int deviceKey[MAX_KEY_LENGTH];

/* Utility function, use to do error checking for CUDA calls
 *
 * Use this function like this:
 *     checkCudaCall(<cuda_call>);
 *
 * For example:
 *     checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));
 *
 * Special case to check the result of the last kernel invocation:
 *     kernel<<<...>>>(...);
 *     checkCudaCall(hipGetLastError());
 **/
static void checkCudaCall(hipError_t result) {
  if (result != hipSuccess) {
    cerr << "cuda error: " << hipGetErrorString(result) << endl;
    exit(EXIT_FAILURE);
  }
}

/* Change this kernel to properly encrypt the given data. The result should be
 * written to the given out data. */
__global__ void encryptKernel(char *deviceDataIn, char *deviceDataOut,
                              int length) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  char input = deviceDataIn[idx];

  if (length == 1) {
    if ((input >= 'A' && input <= 'Z') || (input >= 'a' && input <= 'z')) {
      int shift = deviceKey[0];

      if (input >= 'a' && input <= 'z') {
        // Wrapping alphabet characters formula derived from:
        // https://en.wikipedia.org/wiki/Caesar_cipher
        deviceDataOut[idx] = 'a' + (input - 'a' + shift) % 26;
      } else if (input >= 'A' && input <= 'Z') {
        // Wrapping alphabet characters formula derived from:
        // https://en.wikipedia.org/wiki/Caesar_cipher
        deviceDataOut[idx] = 'A' + (input - 'A' + shift) % 26;
      }
    } else {
      deviceDataOut[idx] = input;
    }
  }

  if (length > 1) {
    if ((input >= 'A' && input <= 'Z') || (input >= 'a' && input <= 'z')) {
      int shift = deviceKey[idx % length];
      if (input >= 'a' && input <= 'z') {
        // Wrapping alphabet characters formula derived from:
        // https://en.wikipedia.org/wiki/Caesar_cipher
        deviceDataOut[idx] = 'a' + (input - 'a' + shift) % 26;
      } else if (input >= 'A' && input <= 'Z') {
        // Wrapping alphabet characters formula derived from:
        // https://en.wikipedia.org/wiki/Caesar_cipher
        deviceDataOut[idx] = 'A' + (input - 'A' + shift) % 26;
      }
    } else {
      deviceDataOut[idx] = input;
    }
  }
}

/* Change this kernel to properly decrypt the given data. The result should be
 * written to the given out data. */
__global__ void decryptKernel(char *deviceDataIn, char *deviceDataOut,
                              int length) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  char input = deviceDataIn[idx];

  if (length == 1) {
    if ((input >= 'A' && input <= 'Z') || (input >= 'a' && input <= 'z')) {
      int shift = deviceKey[0] % 26;
      if (input >= 'a' && input <= 'z') {
        // Wrapping alphabet characters formula derived from:
        // https://en.wikipedia.org/wiki/Caesar_cipher
        deviceDataOut[idx] = 'a' + (input - 'a' - shift + 26) % 26;
      } else if (input >= 'A' && input <= 'Z') {
        // Wrapping alphabet characters formula derived from:
        // https://en.wikipedia.org/wiki/Caesar_cipher
        deviceDataOut[idx] = 'A' + (input - 'A' - shift + 26) % 26;
      }
    } else {
      deviceDataOut[idx] = input;
    }
  }

  if (length >= 1) {
    if ((input >= 'A' && input <= 'Z') || (input >= 'a' && input <= 'z')) {
      int shift = deviceKey[idx % length] % 26;
      if (input >= 'a' && input <= 'z') {
        // Wrapping alphabet characters formula derived from:
        // https://en.wikipedia.org/wiki/Caesar_cipher
        deviceDataOut[idx] = 'a' + (input - 'a' - shift + 26) % 26;
      } else if (input >= 'A' && input <= 'Z') {
        // Wrapping alphabet characters formula derived from:
        // https://en.wikipedia.org/wiki/Caesar_cipher
        deviceDataOut[idx] = 'A' + (input - 'A' - shift + 26) % 26;
      }
    } else {
      deviceDataOut[idx] = input;
    }
  }
}

/* Sequential implementation of encryption with the Shift cipher (and therefore
 * also of Caesar's cipher, if key_length == 1), which you need to implement as
 * well. Then, it can be used to verify your parallel results and compute
 * speedups of your parallelized implementation. */
int EncryptSeq(int n, char *data_in, char *data_out, int key_length, int *key) {
  cout << "ENCRYPT: Key Length = " << key_length << ", Key Values = ";
  for (int i = 0; i < key_length; i++) {
    cout << key[i] << " ";
  }
  cout << endl;

  int i, valid_index = 0;

  timer sequentialTime = timer("Sequential encryption");

  sequentialTime.start();
  for (i = 0; i < n; i++) {
    if (!isalpha(data_in[i])) {
      data_out[i] = data_in[i];
      continue;
    }

    // CAESAR
    if (key_length == 1) {
      if (islower(data_in[i])) {
        data_out[i] = 'a' + ((data_in[i] - 'a' + key[0]) % 26);
      } else if (isupper(data_in[i])) {
        data_out[i] = 'A' + ((data_in[i] - 'A' + key[0]) % 26);
      }
    }
    // VIGENERE
    else {
      int key_index = valid_index % key_length;
      if (islower(data_in[i])) {
        data_out[i] = 'a' + ((data_in[i] - 'a' + key[key_index]) % 26);
      } else if (isupper(data_in[i])) {
        data_out[i] = 'A' + ((data_in[i] - 'A' + key[key_index]) % 26);
      }
      valid_index++;
    }
  }
  sequentialTime.stop();

  cout << fixed << setprecision(6);
  cout << "Encryption (sequential): \t\t" << sequentialTime.getElapsed()
       << " seconds." << endl;

  return 0;
}

/* Sequential implementation of decryption with the Shift cipher (and therefore
 * also of Caesar's cipher, if key_length == 1), which you need to implement as
 * well. Then, it can be used to verify your parallel results and compute
 * speedups of your parallelized implementation. */
int DecryptSeq(int n, char *data_in, char *data_out, int key_length, int *key) {
  cout << "Input to DecryptSeq: ";
  for (int j = 0; j < n; j++) {
    cout << data_in[j];
  }
  cout << endl;

  int i, valid_index = 0;
  timer sequentialTime = timer("Sequential decryption");

  sequentialTime.start();
  for (i = 0; i < n; i++) {
    if (!isalpha(data_in[i])) {
      data_out[i] = data_in[i];
      continue;
    }

    // CAESAR
    if (key_length == 1) {
      int shift = key[0] % 26;
      if (islower(data_in[i])) {
        data_out[i] = 'a' + ((data_in[i] - 'a' - shift + 26) % 26);
      } else if (isupper(data_in[i])) {
        data_out[i] = 'A' + ((data_in[i] - 'A' - shift + 26) % 26);
      }
    }
    // VIGENERE
    else {
      int key_index = valid_index % key_length;
      int shift = key[key_index] % 26;
      if (islower(data_in[i])) {
        data_out[i] = 'a' + ((data_in[i] - 'a' - shift + 26) % 26);
      } else if (isupper(data_in[i])) {
        data_out[i] = 'A' + ((data_in[i] - 'A' - shift + 26) % 26);
      }
      valid_index++;
    }
  }

  sequentialTime.stop();

  cout << fixed << setprecision(6);
  cout << "Decryption (sequential): \t\t" << sequentialTime.getElapsed()
       << " seconds." << endl;

  return 0;
}

/* Wrapper for your encrypt kernel, i.e., does the necessary preparations and
 * calls your kernel. */
int EncryptCuda(int n, char *data_in, char *data_out, int key_length,
                int *key) {
  int threadBlockSize = 4;

  hipMemcpyToSymbol(HIP_SYMBOL(deviceKey), key, (key_length + 1) * sizeof(int));

  // allocate the vectors on the GPU
  char *deviceDataIn = NULL;
  checkCudaCall(hipMalloc((void **)&deviceDataIn, n * sizeof(char)));
  if (deviceDataIn == NULL) {
    cout << "could not allocate memory!" << endl;
    return -1;
  }
  char *deviceDataOut = NULL;
  checkCudaCall(hipMalloc((void **)&deviceDataOut, n * sizeof(char)));
  if (deviceDataOut == NULL) {
    checkCudaCall(hipFree(deviceDataIn));
    cout << "could not allocate memory!" << endl;
    return -1;
  }

  timer kernelTime1 = timer("kernelTime");
  timer memoryTime = timer("memoryTime");

  // copy the original vectors to the GPU
  memoryTime.start();
  checkCudaCall(hipMemcpy(deviceDataIn, data_in, n * sizeof(char),
                           hipMemcpyHostToDevice));
  memoryTime.stop();

  // execute kernel
  kernelTime1.start();

  int gridSize;
  if (n < threadBlockSize) {
    gridSize = 1;
  } else {
    gridSize = (n + threadBlockSize - 1) / threadBlockSize;
  }
  encryptKernel<<<gridSize, threadBlockSize>>>(deviceDataIn, deviceDataOut,
                                               key_length);
  hipDeviceSynchronize();
  kernelTime1.stop();

  // check whether the kernel invocation was successful
  checkCudaCall(hipGetLastError());

  // copy result back
  memoryTime.start();
  checkCudaCall(hipMemcpy(data_out, deviceDataOut, n * sizeof(char),
                           hipMemcpyDeviceToHost));
  memoryTime.stop();

  checkCudaCall(hipFree(deviceDataIn));
  checkCudaCall(hipFree(deviceDataOut));

  cout << fixed << setprecision(6);
  cout << "Encrypt (kernel): \t\t" << kernelTime1.getElapsed() << " seconds."
       << endl;
  cout << "Encrypt (memory): \t\t" << memoryTime.getElapsed() << " seconds."
       << endl;

  return 0;
}

/* Wrapper for your decrypt kernel, i.e., does the necessary preparations and
 * calls your kernel. */
int DecryptCuda(int n, char *data_in, char *data_out, int key_length,
                int *key) {
  int threadBlockSize = 512;

  // allocate the vectors on the GPU
  char *deviceDataIn = NULL;
  checkCudaCall(hipMalloc((void **)&deviceDataIn, n * sizeof(char)));
  if (deviceDataIn == NULL) {
    cout << "could not allocate memory!" << endl;
    return -1;
  }
  char *deviceDataOut = NULL;
  checkCudaCall(hipMalloc((void **)&deviceDataOut, n * sizeof(char)));
  if (deviceDataOut == NULL) {
    checkCudaCall(hipFree(deviceDataIn));
    cout << "could not allocate memory!" << endl;
    return -1;
  }

  timer kernelTime1 = timer("kernelTime");
  timer memoryTime = timer("memoryTime");

  // copy the original vectors to the GPU
  memoryTime.start();
  checkCudaCall(hipMemcpy(deviceDataIn, data_in, n * sizeof(char),
                           hipMemcpyHostToDevice));
  memoryTime.stop();

  // execute kernel
  kernelTime1.start();

  int gridSize;
  if (n < threadBlockSize) {
    gridSize = 1;
  } else {
    gridSize = (n + threadBlockSize - 1) / threadBlockSize;
  }
  decryptKernel<<<gridSize, threadBlockSize>>>(deviceDataIn, deviceDataOut,
                                               key_length);
  hipDeviceSynchronize();
  kernelTime1.stop();

  // check whether the kernel invocation was successful
  checkCudaCall(hipGetLastError());

  // copy result back
  memoryTime.start();
  checkCudaCall(hipMemcpy(data_out, deviceDataOut, n * sizeof(char),
                           hipMemcpyDeviceToHost));
  memoryTime.stop();

  checkCudaCall(hipFree(deviceDataIn));
  checkCudaCall(hipFree(deviceDataOut));

  cout << fixed << setprecision(6);
  cout << "Decrypt (kernel): \t\t" << kernelTime1.getElapsed() << " seconds."
       << endl;
  cout << "Decrypt (memory): \t\t" << memoryTime.getElapsed() << " seconds."
       << endl;

  return 0;
}

/* Entry point to the function! */
int main(int argc, char *argv[]) {
  // Check if there are enough arguments
  if (argc < 2) {
    cout << "Usage: " << argv[0] << " key..." << endl;
    cout << " - key: one or more values for the encryption key, separated "
            "by spaces"
         << endl;

    return EXIT_FAILURE;
  }

  // Parse the keys from the command line arguments
  int key_length = argc - 1;
  int *enc_key = new int[key_length];
  for (int i = 0; i < key_length; i++) {
    enc_key[i] = atoi(argv[i + 1]);
  }

  // Check if the original.data file exists and what it's size is
  int n;
  n = fileSize("original.data");
  if (n == -1) {
    cout << "File not found! Exiting ... " << endl;
    exit(0);
  }

  // Read the file in memory from the disk
  char *data_in = new char[n];
  char *data_out = new char[n];
  readData("original.data", data_in);

  cout << "Encrypting a file of " << n << " characters." << endl;

  EncryptSeq(n, data_in, data_out, key_length, enc_key);
  writeData(n, "sequential.data", data_out);

  EncryptCuda(n, data_in, data_out, key_length, enc_key);
  writeData(n, "cuda.data", data_out);

  readData("cuda.data", data_in);

  cout << "Decrypting a file of " << n << "characters" << endl;
  DecryptSeq(n, data_in, data_out, key_length, enc_key);
  writeData(n, "sequential_recovered.data", data_out);
  DecryptCuda(n, data_in, data_out, key_length, enc_key);
  writeData(n, "recovered.data", data_out);

  delete[] data_in;
  delete[] data_out;

  return 0;
}
