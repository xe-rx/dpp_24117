/*
 * simulate.cu
 *
 * Implementation of a wave equation simulation, parallelized on the GPU using
 * CUDA.
 *
 * You are supposed to edit this file with your implementation, and this file
 * only.
 *
 */

#include <cstdlib>
#include <iostream>

#include "simulate.hh"

using namespace std;


/* Utility function, use to do error checking for CUDA calls
 *
 * Use this function like this:
 *     checkCudaCall(<cuda_call>);
 *
 * For example:
 *     checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));
 * 
 * Special case to check the result of the last kernel invocation:
 *     kernel<<<...>>>(...);
 *     checkCudaCall(hipGetLastError());
**/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(EXIT_FAILURE);
    }
}


/* Function that will simulate the wave equation, parallelized using CUDA.
 *
 * i_max: how many data points are on a single wave
 * t_max: how many iterations the simulation should run
 * num_threads: how many threads to use (excluding the main threads)
 * old_array: array of size i_max filled with data for t-1
 * current_array: array of size i_max filled with data for t
 * next_array: array of size i_max. You should fill this with t+1
 * 
 */
double *simulate(const long i_max, const long t_max, const long block_size,
                 double *old_array, double *current_array, double *next_array) {

    // YOUR CODE HERE

    return current_array;
}
